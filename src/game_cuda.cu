#include "hip/hip_runtime.h"
// To compile: make cuda
// To run: ./a.out [width] [height] [input_file]

#define BLOCK_SIZE 32

#define GEN_LIMIT 1000

#define CHECK_SIMILARITY
#define SIMILARITY_FREQUENCY 3

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hip/hip_runtime.h>

#include "../include/timestamp.h"

#define cudaSafeCall(call)                                                \
    {                                                                     \
        hipError_t err = call;                                             \
        if (hipSuccess != err)                                           \
        {                                                                 \
            fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n", \
                    __FILE__, __LINE__, hipGetErrorString(err));         \
            exit(EXIT_FAILURE);                                           \
        }                                                                 \
    }

void perror_exit(const char *message)
{
    perror(message);
    exit(EXIT_FAILURE);
}

void print_to_file(unsigned char *univ, int width, int height)
{
    FILE *fout = fopen("./cuda_output.out", "w"); // printing the result to a file with
                                                  // 1 or 0 (1 being an alive cell and 0 a dead cell)
    for (int i = 1; i <= width; i++)
    {
        for (int j = 1; j <= height; j++)
        {
            fprintf(fout, univ[i * (width + 2) + j] ? "1" : "0");
        }
        fprintf(fout, "\n");
    }

    fflush(fout);
    fclose(fout);
}

__global__ void halo_rows(unsigned char *univ, int height)
{
    // Copy the actual rows from both sides to the additional halo rows in the array
    int index = blockDim.x * blockIdx.x + threadIdx.x + 1;

    if (index <= height)
    {
        univ[(height + 2) * (height + 1) + index] = univ[(height + 2) + index];
        univ[index] = univ[(height + 2) * height + index];
    }
}

__global__ void halo_cols(unsigned char *univ, int width)
{
    // Copy the actual columns from both sides to the additional halo columns in the array
    int index = blockDim.x * blockIdx.x + threadIdx.x;

    if (index <= width + 1)
    {
        univ[index * (width + 2) + width + 1] = univ[index * (width + 2) + 1];
        univ[index * (width + 2)] = univ[index * (width + 2) + width];
    }
}

__global__ void compare(unsigned char *univ, unsigned char *new_univ, long long int size, int *same)
{
    // Use reduction to check if two arrays are the same by adding 1 to a counter for each common cell:
    // If it's (width + 2) * (height + 2), then they are identical
    __shared__ int s_array[BLOCK_SIZE];

    int index = threadIdx.x;
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    s_array[index] = (i < size) ? (int)(univ[i] == new_univ[i]) : 0;

    __syncthreads();

    for (int i = blockDim.x / 2; i > 0; i >>= 1)
    {
        if (index < i)
        {
            s_array[index] += s_array[index + i];
        }
        __syncthreads();
    }

    if (index == 0)
        atomicAdd(same, s_array[0]);
}

__global__ void empty(unsigned char *temp_univ, long long int size, int *alive)
{
    // Use reduction to check if the array is empty by adding all values in the array:
    // If it's zero, then there is no alive cell in the array
    __shared__ int s_array[BLOCK_SIZE];

    int index = threadIdx.x;
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    s_array[index] = (i < size) ? (int)temp_univ[i] : 0;

    __syncthreads();

    for (int i = blockDim.x / 2; i > 0; i >>= 1)
    {
        if (index < i)
        {
            s_array[index] += s_array[index + i];
        }
        __syncthreads();
    }

    if (index == 0)
        atomicAdd(alive, s_array[0]);
}

__global__ void evolve(unsigned char *univ, unsigned char *new_univ, int width, int height)
{
    // Generate new generation: keep it in new_univ
    int iy = blockDim.y * blockIdx.y + threadIdx.y + 1;
    int ix = blockDim.x * blockIdx.x + threadIdx.x + 1;
    int id = iy * (width + 2) + ix;

    int neighbors = 0;

    if (iy <= height && ix <= width)
    {
        neighbors = univ[id + (height + 2)] +                           // Upper neighbor
                    univ[id - (height + 2)] +                           // Lower neighbor
                    univ[id + 1] +                                      // Right neighbor
                    univ[id - 1] +                                      // Left neighbor
                    univ[id + (height + 3)] + univ[id - (height + 3)] + // Diagonal neighbors
                    univ[id - (height + 1)] + univ[id + (height + 1)];

        new_univ[id] = (neighbors == 3 || (neighbors == 2 && univ[id]));
    }
}

int main(int argc, char *argv[])
{
    int width = 0, height = 0;
    long long int size = 0;

    width = atoi(argv[1]);
    height = atoi(argv[2]);
    size = (width + 2) * (height + 2);

    // Allocate space for the game array
    unsigned char *univ = (unsigned char *)calloc(size, sizeof(unsigned char));
    if (univ == NULL)
        perror_exit("calloc: ");

    // Fetch all the values from an input file
    FILE *filePtr = fopen(argv[3], "r");
    if (filePtr == NULL)
        perror_exit("fopen: ");

    for (int i = 1; i <= width; i++)
    {
        for (int j = 1; j <= height;)
        {
            char c = fgetc(filePtr);
            if ((c != EOF) && (c != '\n'))
            {
                univ[i * (width + 2) + j] = c - 48;
                j++;
            }
        }
    }
    fclose(filePtr);
    filePtr = NULL;

    // Allocate two game arrays in the GPU memory and transfer the contents of the original array
    unsigned char *d_univ, *d_new_univ;

    cudaSafeCall(hipMalloc((void **)&d_univ, size * sizeof(unsigned char)));
    cudaSafeCall(hipMalloc((void **)&d_new_univ, size * sizeof(unsigned char)));

    cudaSafeCall(hipMemcpy(d_univ, univ, size * sizeof(unsigned char), hipMemcpyHostToDevice));
    cudaSafeCall(hipMemcpy(d_new_univ, univ, size * sizeof(unsigned char), hipMemcpyHostToDevice));

    // Allocate space in the GPU memory for two counters/flags: alive to check is the array is
    // empty and the same to compare two generations
    int alive = 0, *d_alive,
        same = 0, *d_same;

    cudaSafeCall(hipMalloc((void **)&d_alive, sizeof(int)));
    cudaSafeCall(hipMalloc((void **)&d_same, sizeof(int)));

    cudaSafeCall(hipMemcpy(d_alive, &alive, sizeof(int), hipMemcpyHostToDevice));
    cudaSafeCall(hipMemcpy(d_same, &same, sizeof(int), hipMemcpyHostToDevice));

    // Set up parameters for number of blocks and threads per block for kernel launches
    dim3 block_size(BLOCK_SIZE, BLOCK_SIZE, 1);
    int linGrid = (int)ceil(width / (float)BLOCK_SIZE);
    dim3 grid_size(linGrid, linGrid, 1);

    int rows_grid_size = (int)ceil(width / (float)BLOCK_SIZE);
    int cols_grid_size = (int)ceil((width + 2) / (float)BLOCK_SIZE);
    int univ_grid_size = (width <= BLOCK_SIZE * 2 || height <= BLOCK_SIZE) ? (int)ceil((width + 2) * (height + 2) / (float)BLOCK_SIZE) : (int)ceil((width + 2) * (height + 2) / (float)(BLOCK_SIZE * BLOCK_SIZE));

    int generation = 0;
#ifdef CHECK_SIMILARITY
    int counter = 0;
#endif

    // Get currect timestamp: calculations are about to start
    timestamp t_start;
    t_start = getTimestamp();

    while (generation < GEN_LIMIT)
    {
        // Create halo rows and columns for neighbors
        halo_rows<<<rows_grid_size, BLOCK_SIZE>>>(d_univ, height);
        cudaSafeCall(hipGetLastError());
        cudaSafeCall(hipDeviceSynchronize());

        halo_cols<<<cols_grid_size, BLOCK_SIZE>>>(d_univ, width);
        cudaSafeCall(hipGetLastError());
        cudaSafeCall(hipDeviceSynchronize());

        // Generate a new generations from d_univ and store in on d_new_univ
        evolve<<<grid_size, block_size>>>(d_univ, d_new_univ, width, height);
        cudaSafeCall(hipGetLastError());
        cudaSafeCall(hipDeviceSynchronize());

#ifdef CHECK_SIMILARITY
        // Check new and old generation, if they are the same: exit
        counter++;
        if (counter == SIMILARITY_FREQUENCY)
        {
            compare<<<univ_grid_size, BLOCK_SIZE>>>(d_univ, d_new_univ, size, d_same);
            cudaSafeCall(hipGetLastError());
            cudaSafeCall(hipDeviceSynchronize());

            cudaSafeCall(hipMemcpy(&same, d_same, sizeof(int), hipMemcpyDeviceToHost));

            if (same == size)
                break;
            else
                cudaSafeCall(hipMemset(d_same, 0, sizeof(int)));

            counter = 0;
        }
#endif

        // Check if the array is empty, if it is: exit
        empty<<<univ_grid_size, BLOCK_SIZE>>>(d_new_univ, size, d_alive);
        cudaSafeCall(hipGetLastError());
        cudaSafeCall(hipDeviceSynchronize());

        cudaSafeCall(hipMemcpy(&alive, d_alive, sizeof(int), hipMemcpyDeviceToHost));

        if (alive == 0)
            break;
        else
            cudaSafeCall(hipMemset(d_alive, 0, sizeof(int)));

        // Pointer switch for fast array switching: d_univ will be used in the next loop
        unsigned char *temp_univ = d_univ;
        d_univ = d_new_univ;
        d_new_univ = temp_univ;

        generation++;
    }

    // Get the total duration of the loop above in milliseconds
    float msecs = getElapsedtime(t_start);

    cudaSafeCall(hipMemcpy(univ, d_univ, size * sizeof(unsigned char), hipMemcpyDeviceToHost));
    print_to_file(univ, width, height);

    // Free allocated memory
    cudaSafeCall(hipFree(d_univ));
    cudaSafeCall(hipFree(d_new_univ));

    cudaSafeCall(hipFree(d_alive));
    cudaSafeCall(hipFree(d_same));

    free(univ);
    univ = NULL;

    printf("Generations:\t%d\n", generation);
    printf("Execution time:\t%.2f msecs\n", msecs);

    printf("Finished\n");
    fflush(stdout);

    return 0;
}
